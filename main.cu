#include<iostream>
#include<vector>
#include<source_location>
#include<fstream>
#include<hip/hip_runtime.h>



__constant__ float fdtd_coeff[4]={1225./1024, 245./3072, 49./5120, 5./7168};


void CHECK_CALL(){
	hipError_t err = hipGetLastError();
	if(err  != hipSuccess){
		//std::source_location location = std::source_location::current();
		std::cout << "Cuda error: " << err << std::endl;
			/*":"
			<< location.file_name() << ":"
			<< location.line() << std::endl;*/
	}
}


__global__
void kernel_dPdt(float *P1, float *P_1, float *Vx, float *Vy, float *vel, float *rho,
				  int width, int depth, float dx, float dt){

	int ix = threadIdx.x + threadIdx.x * blockIdx.x;
	int iy = threadIdx.y + threadIdx.y * blockIdx.y;
	int tid = ix + iy * width;

	if(ix > 3 && ix < (width - 3) && iy > 3 && iy < (depth - 3)){
		P1[tid] += -1 * dt * dt * vel[tid] * vel[tid] * rho[tid] * (
			fdtd_coeff[0] * (Vx[tid] - Vx[tid-1] + Vy[tid] - Vy[tid-1*width]) -
			fdtd_coeff[1] * (Vx[tid+1] - Vx[tid-2] + Vy[tid+1*width] - Vy[tid-2*width]) +
			fdtd_coeff[2] * (Vx[tid+2] - Vx[tid-3] + Vy[tid+2*width] - Vy[tid-3*width]) -
			fdtd_coeff[3] * (Vx[tid+3] - Vx[tid-4] + Vy[tid+3*width] - Vy[tid-4*width]) ) / dx;
	}
}


__global__
void kernel_dVdt(float *P, float *Vx, float *Vy, float *rho,
				  int width, int depth, float dx, float dy, float dt){

	int ix = threadIdx.x + threadIdx.x * blockIdx.x;
	int iy = threadIdx.y + threadIdx.y * blockIdx.y;
	int tid = ix + iy * width;

	if(ix > 2 && ix < (width - 4) && iy < depth){
		Vx[tid] += -1 * (2. / (rho[tid] + rho[tid+1])) * dt * (
			fdtd_coeff[0] * (P[tid+1] - P[tid]) - 
			fdtd_coeff[1] * (P[tid+2] - P[tid-1]) +
			fdtd_coeff[2] * (P[tid+3] - P[tid-2]) -
			fdtd_coeff[3] * (P[tid+4] - P[tid-3]) ) / dx;
	}
	
	if(iy > 2 && iy < (depth - 4) && ix < width){
		Vy[tid] += -1 * (2. / (rho[tid] + rho[tid+1*width])) * dt * (
			fdtd_coeff[0] * (P[tid+1*width] - P[tid]) - 
			fdtd_coeff[1] * (P[tid+2*width] - P[tid-1*width]) +
			fdtd_coeff[2] * (P[tid+3*width] - P[tid-2*width]) -
			fdtd_coeff[3] * (P[tid+4*width] - P[tid-3*width]) ) / dy;
	}
}


__global__
void kernel_add_source(float *P, float *source, int time_sample, int sloc_x, int sloc_y,
					   int width){
	int ix = threadIdx.x;
	P[ix + sloc_x + width * sloc_y] += 1.;
}


void save_wavefield(float *P_h, float *P_d, int width, int depth, int it){

	hipMemcpy(P_h, P_d, width * depth * sizeof(float), hipMemcpyDeviceToHost);
	std::ofstream out_stream("wavefield.bin", std::ios::binary);
	out_stream.seekp(it*width*depth*sizeof(float));

}


void propagate(float *P1, float *P_1, float *Vx, float *Vy, float *rho, float *vel,
			   float dx, float dy, float dt, int width, int depth, int time_samples){

	dim3 block_size(16,16);
	dim3 grid_size(width/16+1, depth/16+1);

	for(int c=0;c<time_samples;c++){
		std::cout << "iteration: " << c << std::endl;
		kernel_dVdt<<<grid_size, block_size>>>(P1,Vx,Vy,rho,width,depth,dx,dy,dt);
		CHECK_CALL();
		kernel_dPdt<<<grid_size, block_size>>>(P1,P_1,Vx,Vy,vel,rho,width,depth,dx,dt);
		CHECK_CALL();
		kernel_add_source<<<1,1>>>(P1,P1,c,width/2,depth/2,width);
		CHECK_CALL();
	}
}


int main(){

	int model_width = 1024;
	int model_depth = 1024;

	float dx = 12.5;
	float dy = 12.5;
	float dt = 1e-3;
	float total_time = 3.0;
	int time_samples = total_time / dt + 1;

	std::vector<float> vel_model_h = std::vector<float>(model_width * model_depth);
	std::vector<float> rho_model_h = std::vector<float>(model_width * model_depth);

	float *vel_model_d, *rho_model_d;
	float *P1, *P_1, *dP_dx, *dP_dy, *Vx, *Vy, *dVx_dt, *dVy_dt, *dVx_dx, *dVy_dy;
	hipMalloc(&vel_model_d, model_width * model_depth * sizeof(float));
	hipMalloc(&rho_model_d, model_width * model_depth * sizeof(float));
	hipMalloc(&P1, model_width * model_depth * sizeof(float));
	hipMalloc(&Vx, model_width * model_depth * sizeof(float));
	hipMalloc(&Vy, model_width * model_depth * sizeof(float));
	hipMalloc(&P_1, model_width * model_depth * sizeof(float));
	hipMalloc(&dP_dx, model_width * model_depth * sizeof(float));
	hipMalloc(&dP_dy, model_width * model_depth * sizeof(float));
	hipMalloc(&dVx_dt, model_width * model_depth * sizeof(float));
	hipMalloc(&dVx_dx, model_width * model_depth * sizeof(float));
	hipMalloc(&dVy_dt, model_width * model_depth * sizeof(float));
	hipMalloc(&dVy_dy, model_width * model_depth * sizeof(float));

	for(int c=0;c<model_width*model_depth;c++){
		vel_model_h[c] = 1500.;
		rho_model_h[c] = 2600.;
	}

	hipMemcpy(vel_model_d, vel_model_h.data(), model_width * model_depth * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(rho_model_d, rho_model_h.data(), model_width * model_depth * sizeof(float), hipMemcpyHostToDevice);

	propagate(P1,P_1,Vx,Vy,rho_model_d,vel_model_d,dx,dy,dt,model_width,model_depth,time_samples);

	hipFree(vel_model_d);
	hipFree(rho_model_d);
	hipFree(P1);
	hipFree(Vx);
	hipFree(Vy);
	hipFree(P_1);
	hipFree(dP_dx);
	hipFree(dP_dy);
	hipFree(dVx_dt);
	hipFree(dVx_dx);
	hipFree(dVy_dt);
	hipFree(dVy_dy);

	return 0;
}
